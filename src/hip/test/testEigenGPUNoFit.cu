#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <iostream>

#include <Eigen/Core>
#include <Eigen/Eigenvalues>

#include "CUDACore/cudaCheck.h"
#include "CUDACore/requireDevices.h"
#include "test_common.h"

using namespace Eigen;

using Matrix5d = Matrix<double, 5, 5>;

__host__ __device__ void eigenValues(Matrix3d *m, Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType *ret) {
#if TEST_DEBUG
  printf("Matrix(0,0): %f\n", (*m)(0, 0));
  printf("Matrix(1,1): %f\n", (*m)(1, 1));
  printf("Matrix(2,2): %f\n", (*m)(2, 2));
#endif
  SelfAdjointEigenSolver<Matrix3d> es;
  es.computeDirect(*m);
  (*ret) = es.eigenvalues();
  return;
}

__global__ void kernel(Matrix3d *m, Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType *ret) {
  eigenValues(m, ret);
}

__global__ void kernelInverse3x3(Matrix3d *in, Matrix3d *out) { (*out) = in->inverse(); }

__global__ void kernelInverse4x4(Matrix4d *in, Matrix4d *out) { (*out) = in->inverse(); }

__global__ void kernelInverse5x5(Matrix5d *in, Matrix5d *out) { (*out) = in->inverse(); }

template <typename M1, typename M2, typename M3>
__global__ void kernelMultiply(M1 *J, M2 *C, M3 *result) {
//  Map<M3> res(result->data());
#if TEST_DEBUG
  printf("*** GPU IN ***\n");
#endif
  printIt(J);
  printIt(C);
  //  res.noalias() = (*J) * (*C);
  //  printIt(&res);
  (*result) = (*J) * (*C);
#if TEST_DEBUG
  printf("*** GPU OUT ***\n");
#endif
  return;
}

template <int row1, int col1, int row2, int col2>
void testMultiply() {
  std::cout << "TEST MULTIPLY" << std::endl;
  std::cout << "Product of type " << row1 << "x" << col1 << " * " << row2 << "x" << col2 << std::endl;
  Eigen::Matrix<double, row1, col1> J;
  fillMatrix(J);
  Eigen::Matrix<double, row2, col2> C;
  fillMatrix(C);
  Eigen::Matrix<double, row1, col2> multiply_result = J * C;
#if TEST_DEBUG
  std::cout << "Input J:" << std::endl;
  printIt(&J);
  std::cout << "Input C:" << std::endl;
  printIt(&C);
  std::cout << "Output:" << std::endl;
  printIt(&multiply_result);
#endif
  // GPU
  Eigen::Matrix<double, row1, col1> *JGPU = nullptr;
  Eigen::Matrix<double, row2, col2> *CGPU = nullptr;
  Eigen::Matrix<double, row1, col2> *multiply_resultGPU = nullptr;
  Eigen::Matrix<double, row1, col2> *multiply_resultGPUret = new Eigen::Matrix<double, row1, col2>();

  cudaCheck(hipMalloc((void **)&JGPU, sizeof(Eigen::Matrix<double, row1, col1>)));
  cudaCheck(hipMalloc((void **)&CGPU, sizeof(Eigen::Matrix<double, row2, col2>)));
  cudaCheck(hipMalloc((void **)&multiply_resultGPU, sizeof(Eigen::Matrix<double, row1, col2>)));
  cudaCheck(hipMemcpy(JGPU, &J, sizeof(Eigen::Matrix<double, row1, col1>), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(CGPU, &C, sizeof(Eigen::Matrix<double, row2, col2>), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(
      multiply_resultGPU, &multiply_result, sizeof(Eigen::Matrix<double, row1, col2>), hipMemcpyHostToDevice));

  hipLaunchKernelGGL(kernelMultiply, dim3(1), dim3(1), 0, 0, JGPU, CGPU, multiply_resultGPU);
  cudaCheck(hipDeviceSynchronize());

  cudaCheck(hipMemcpy(
      multiply_resultGPUret, multiply_resultGPU, sizeof(Eigen::Matrix<double, row1, col2>), hipMemcpyDeviceToHost));
  printIt(multiply_resultGPUret);
  assert(isEqualFuzzy(multiply_result, (*multiply_resultGPUret)));
}

void testInverse3x3() {
  std::cout << "TEST INVERSE 3x3" << std::endl;
  Matrix3d m;
  fillMatrix(m);
  m += m.transpose().eval();

  Matrix3d m_inv = m.inverse();
  Matrix3d *mGPU = nullptr;
  Matrix3d *mGPUret = nullptr;
  Matrix3d *mCPUret = new Matrix3d();

#if TEST_DEBUG
  std::cout << "Here is the matrix m:" << std::endl << m << std::endl;
  std::cout << "Its inverse is:" << std::endl << m.inverse() << std::endl;
#endif
  cudaCheck(hipMalloc((void **)&mGPU, sizeof(Matrix3d)));
  cudaCheck(hipMalloc((void **)&mGPUret, sizeof(Matrix3d)));
  cudaCheck(hipMemcpy(mGPU, &m, sizeof(Matrix3d), hipMemcpyHostToDevice));

  hipLaunchKernelGGL(kernelInverse3x3, dim3(1), dim3(1), 0, 0, mGPU, mGPUret);
  cudaCheck(hipDeviceSynchronize());

  cudaCheck(hipMemcpy(mCPUret, mGPUret, sizeof(Matrix3d), hipMemcpyDeviceToHost));
#if TEST_DEBUG
  std::cout << "Its GPU inverse is:" << std::endl << (*mCPUret) << std::endl;
#endif
  assert(isEqualFuzzy(m_inv, *mCPUret));
}

void testInverse4x4() {
  std::cout << "TEST INVERSE 4x4" << std::endl;
  Matrix4d m;
  fillMatrix(m);
  m += m.transpose().eval();

  Matrix4d m_inv = m.inverse();
  Matrix4d *mGPU = nullptr;
  Matrix4d *mGPUret = nullptr;
  Matrix4d *mCPUret = new Matrix4d();

#if TEST_DEBUG
  std::cout << "Here is the matrix m:" << std::endl << m << std::endl;
  std::cout << "Its inverse is:" << std::endl << m.inverse() << std::endl;
#endif
  cudaCheck(hipMalloc((void **)&mGPU, sizeof(Matrix4d)));
  cudaCheck(hipMalloc((void **)&mGPUret, sizeof(Matrix4d)));
  cudaCheck(hipMemcpy(mGPU, &m, sizeof(Matrix4d), hipMemcpyHostToDevice));

  hipLaunchKernelGGL(kernelInverse4x4, dim3(1), dim3(1), 0, 0, mGPU, mGPUret);
  cudaCheck(hipDeviceSynchronize());

  cudaCheck(hipMemcpy(mCPUret, mGPUret, sizeof(Matrix4d), hipMemcpyDeviceToHost));
#if TEST_DEBUG
  std::cout << "Its GPU inverse is:" << std::endl << (*mCPUret) << std::endl;
#endif
  assert(isEqualFuzzy(m_inv, *mCPUret));
}

void testInverse5x5() {
  std::cout << "TEST INVERSE 5x5" << std::endl;
  Matrix5d m;
  fillMatrix(m);
  m += m.transpose().eval();

  Matrix5d m_inv = m.inverse();
  Matrix5d *mGPU = nullptr;
  Matrix5d *mGPUret = nullptr;
  Matrix5d *mCPUret = new Matrix5d();

#if TEST_DEBUG
  std::cout << "Here is the matrix m:" << std::endl << m << std::endl;
  std::cout << "Its inverse is:" << std::endl << m.inverse() << std::endl;
#endif
  cudaCheck(hipMalloc((void **)&mGPU, sizeof(Matrix5d)));
  cudaCheck(hipMalloc((void **)&mGPUret, sizeof(Matrix5d)));
  cudaCheck(hipMemcpy(mGPU, &m, sizeof(Matrix5d), hipMemcpyHostToDevice));

  hipLaunchKernelGGL(kernelInverse5x5, dim3(1), dim3(1), 0, 0, mGPU, mGPUret);
  cudaCheck(hipDeviceSynchronize());

  cudaCheck(hipMemcpy(mCPUret, mGPUret, sizeof(Matrix5d), hipMemcpyDeviceToHost));
#if TEST_DEBUG
  std::cout << "Its GPU inverse is:" << std::endl << (*mCPUret) << std::endl;
#endif
  assert(isEqualFuzzy(m_inv, *mCPUret));
}

void testEigenvalues() {
  std::cout << "TEST EIGENVALUES" << std::endl;
  Matrix3d m;
  fillMatrix(m);
  m += m.transpose().eval();

  Matrix3d *m_gpu = nullptr;
  Matrix3d *mgpudebug = new Matrix3d();
  Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType *ret =
      new Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType;
  Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType *ret1 =
      new Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType;
  Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType *ret_gpu = nullptr;
  eigenValues(&m, ret);
#if TEST_DEBUG
  std::cout << "Generated Matrix M 3x3:\n" << m << std::endl;
  std::cout << "The eigenvalues of M are:" << std::endl << (*ret) << std::endl;
  std::cout << "*************************\n\n" << std::endl;
#endif
  cudaCheck(hipMalloc((void **)&m_gpu, sizeof(Matrix3d)));
  cudaCheck(hipMalloc((void **)&ret_gpu, sizeof(Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType)));
  cudaCheck(hipMemcpy(m_gpu, &m, sizeof(Matrix3d), hipMemcpyHostToDevice));

  hipLaunchKernelGGL(kernel, dim3(1), dim3(1), 0, 0, m_gpu, ret_gpu);
  cudaCheck(hipDeviceSynchronize());

  cudaCheck(hipMemcpy(mgpudebug, m_gpu, sizeof(Matrix3d), hipMemcpyDeviceToHost));
  cudaCheck(
      hipMemcpy(ret1, ret_gpu, sizeof(Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType), hipMemcpyDeviceToHost));
#if TEST_DEBUG
  std::cout << "GPU Generated Matrix M 3x3:\n" << (*mgpudebug) << std::endl;
  std::cout << "GPU The eigenvalues of M are:" << std::endl << (*ret1) << std::endl;
  std::cout << "*************************\n\n" << std::endl;
#endif
  assert(isEqualFuzzy(*ret, *ret1));
}

int main(int argc, char *argv[]) {
  cms::hiptest::requireDevices();

  testEigenvalues();
  testInverse3x3();
  testInverse4x4();
  testInverse5x5();

  testMultiply<1, 2, 2, 1>();
  testMultiply<1, 2, 2, 2>();
  testMultiply<1, 2, 2, 3>();
  testMultiply<1, 2, 2, 4>();
  testMultiply<1, 2, 2, 5>();
  testMultiply<2, 1, 1, 2>();
  testMultiply<2, 1, 1, 3>();
  testMultiply<2, 1, 1, 4>();
  testMultiply<2, 1, 1, 5>();
  testMultiply<2, 2, 2, 2>();
  testMultiply<2, 3, 3, 1>();
  testMultiply<2, 3, 3, 2>();
  testMultiply<2, 3, 3, 4>();
  testMultiply<2, 3, 3, 5>();
  testMultiply<3, 2, 2, 3>();
  testMultiply<2, 3, 3, 3>();  // DOES NOT COMPILE W/O PATCHING EIGEN
  testMultiply<3, 3, 3, 3>();
  testMultiply<8, 8, 8, 8>();
  testMultiply<3, 4, 4, 3>();
  testMultiply<2, 4, 4, 2>();
  testMultiply<3, 4, 4, 2>();  // DOES NOT COMPILE W/O PATCHING EIGEN

  return 0;
}
