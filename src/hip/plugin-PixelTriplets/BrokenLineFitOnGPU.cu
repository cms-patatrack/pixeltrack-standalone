#include "hip/hip_runtime.h"
#include "BrokenLineFitOnGPU.h"
#include "CUDACore/device_unique_ptr.h"

void HelixFitOnGPU::launchBrokenLineKernels(HitsView const *hv,
                                            uint32_t hitsInFit,
                                            uint32_t maxNumberOfTuples,
                                            hipStream_t stream) {
  assert(tuples_d);

  auto blockSize = 64;
  auto numberOfBlocks = (maxNumberOfConcurrentFits_ + blockSize - 1) / blockSize;

  //  Fit internals
  auto hitsGPU_ = cms::hip::make_device_unique<double[]>(
      maxNumberOfConcurrentFits_ * sizeof(Rfit::Matrix3xNd<4>) / sizeof(double), stream);
  auto hits_geGPU_ = cms::hip::make_device_unique<float[]>(
      maxNumberOfConcurrentFits_ * sizeof(Rfit::Matrix6x4f) / sizeof(float), stream);
  auto fast_fit_resultsGPU_ = cms::hip::make_device_unique<double[]>(
      maxNumberOfConcurrentFits_ * sizeof(Rfit::Vector4d) / sizeof(double), stream);

  for (uint32_t offset = 0; offset < maxNumberOfTuples; offset += maxNumberOfConcurrentFits_) {
    // fit triplets
    hipLaunchKernelGGL(HIP_KERNEL_NAME(kernelBLFastFit<3>),
                       dim3(numberOfBlocks),
                       dim3(blockSize),
                       0,
                       stream,
                       tuples_d,
                       tupleMultiplicity_d,
                       hv,
                       hitsGPU_.get(),
                       hits_geGPU_.get(),
                       fast_fit_resultsGPU_.get(),
                       3,
                       offset);
    cudaCheck(hipGetLastError());

    hipLaunchKernelGGL(HIP_KERNEL_NAME(kernelBLFit<3>),
                       dim3(numberOfBlocks),
                       dim3(blockSize),
                       0,
                       stream,
                       tupleMultiplicity_d,
                       bField_,
                       outputSoa_d,
                       hitsGPU_.get(),
                       hits_geGPU_.get(),
                       fast_fit_resultsGPU_.get(),
                       3,
                       offset);
    cudaCheck(hipGetLastError());

    // fit quads
    hipLaunchKernelGGL(HIP_KERNEL_NAME(kernelBLFastFit<4>),
                       dim3(numberOfBlocks / 4),
                       dim3(blockSize),
                       0,
                       stream,
                       tuples_d,
                       tupleMultiplicity_d,
                       hv,
                       hitsGPU_.get(),
                       hits_geGPU_.get(),
                       fast_fit_resultsGPU_.get(),
                       4,
                       offset);
    cudaCheck(hipGetLastError());

    hipLaunchKernelGGL(HIP_KERNEL_NAME(kernelBLFit<4>),
                       dim3(numberOfBlocks / 4),
                       dim3(blockSize),
                       0,
                       stream,
                       tupleMultiplicity_d,
                       bField_,
                       outputSoa_d,
                       hitsGPU_.get(),
                       hits_geGPU_.get(),
                       fast_fit_resultsGPU_.get(),
                       4,
                       offset);
    cudaCheck(hipGetLastError());

    if (fit5as4_) {
      // fit penta (only first 4)
      hipLaunchKernelGGL(HIP_KERNEL_NAME(kernelBLFastFit<4>),
                         dim3(numberOfBlocks / 4),
                         dim3(blockSize),
                         0,
                         stream,
                         tuples_d,
                         tupleMultiplicity_d,
                         hv,
                         hitsGPU_.get(),
                         hits_geGPU_.get(),
                         fast_fit_resultsGPU_.get(),
                         5,
                         offset);
      cudaCheck(hipGetLastError());

      hipLaunchKernelGGL(HIP_KERNEL_NAME(kernelBLFit<4>),
                         dim3(numberOfBlocks / 4),
                         dim3(blockSize),
                         0,
                         stream,
                         tupleMultiplicity_d,
                         bField_,
                         outputSoa_d,
                         hitsGPU_.get(),
                         hits_geGPU_.get(),
                         fast_fit_resultsGPU_.get(),
                         5,
                         offset);
      cudaCheck(hipGetLastError());
    } else {
      // fit penta (all 5)
      hipLaunchKernelGGL(HIP_KERNEL_NAME(kernelBLFastFit<5>),
                         dim3(numberOfBlocks / 4),
                         dim3(blockSize),
                         0,
                         stream,
                         tuples_d,
                         tupleMultiplicity_d,
                         hv,
                         hitsGPU_.get(),
                         hits_geGPU_.get(),
                         fast_fit_resultsGPU_.get(),
                         5,
                         offset);
      cudaCheck(hipGetLastError());

      hipLaunchKernelGGL(HIP_KERNEL_NAME(kernelBLFit<5>),
                         dim3(numberOfBlocks / 4),
                         dim3(blockSize),
                         0,
                         stream,
                         tupleMultiplicity_d,
                         bField_,
                         outputSoa_d,
                         hitsGPU_.get(),
                         hits_geGPU_.get(),
                         fast_fit_resultsGPU_.get(),
                         5,
                         offset);
      cudaCheck(hipGetLastError());
    }

  }  // loop on concurrent fits
}
